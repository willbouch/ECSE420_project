#include "hip/hip_runtime.h"
#include "gputimer.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>


char CHARS_MODE_0[] = {'4', '5', '2', '3', '6', '8', '1', '9', '0', '7'};
#define LENGTH_MODE_0 10

char CHARS_MODE_1[] = {'z', 'w', '4', '7', 'o', '8', '0', 'i', 'r', 's', 'x', '2', '5', 'y', 'q', 'g', '3', 'v', 'c', 'l', 'n', 'u', 'j', '6', '1', 'e', 'b', 't', 'd', 'f', 'a', 'k', 'm', '9', 'p', 'h'};

#define LENGTH_MODE_1 36

char CHARS_MODE_2[] = {'0', 'X', 'D', 'L', 'z', 'y', 'e', 'd', 's', 'R', 'n', 'm', '1', 'B', '8', 'F', 'w', 'V', 'M', 'x', '2', 'j', '4', 'E', 'H', 'a', 'W', 'Z', 'b', 'o', 'r', 'A', 'u', 'U', 'I', 'T', 'q', 'c', 't', 'l', 'g', 'K', '7', '6', 'S', '9', 'O', 'f', 'P', 'J', 'k', '3', 'C', '5', 'v', 'Y', 'p', 'h', 'i', 'Q', 'G', 'N'};

#define LENGTH_MODE_2 62

char CHARS_MODE_3[] = {'m', 'R', 'J', 'D', '#', '6', '9', 'T', 'l', '1', 'V', 'p', 'N', 'i', 'v', 'h', 'q', '*', 'd', 'o', 'B', '3', '4', 'x', 'M', 'r', 'C', 'S', '7', 'u', 'A', 'y', '&', 'f', 'P', 'b', '8', 'z', 'X', 'w', 'E', 'Y', 'j', 'I', 'L', 'g', 'W', '!', '2', 'c', 'n', 'G', '0', 'O', 't', 'K', 'a', '$', 'e', 's', 'H', '?', 'k', 'U', 'Z', '5', 'Q', 'F'};

#define LENGTH_MODE_3 68

//https://stackoverflow.com/questions/19600879/how-to-compare-arrays-of-char-in-cuda-c/19602075#19602075
__device__ int cuda_strcmp(const char* str_a, const char* str_b, int len) {
    int match = 0;
    unsigned i = 0;
    while ((i < len) && (match == 0)) {
        if (str_a[i] != str_b[i]) {
            match = i + 1;
        }
        i++;
    }
    return match;
}

__device__ char *generatePassword(int pwLength, char *chars, int charsLength, int seed)
{
    int quotient = seed;
    int remainder;
    char *pw = (char *)malloc(pwLength * sizeof(char));
    for (int i = pwLength - 1; i >= 0; i--)
    {
        remainder = quotient % charsLength;
        quotient = quotient / charsLength;
        char c = chars[remainder];
        pw[i] = c;
    }
    pw[pwLength] = '\n';
    return pw;
}

__global__ void bruteForce(char *truePassword, int pwLength, char *chars, int charsLength, volatile int *found, char *foundPassword, long maxIndex)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int threadCnt = blockDim.x * gridDim.x;

    for (int i = id; i < maxIndex; i += threadCnt)
    {
        if (*found) break;
        char *testPassword = generatePassword(pwLength, chars, charsLength, i);
        if (cuda_strcmp(truePassword, testPassword, pwLength) == 0)
        {
            *found = 1;
            memcpy(foundPassword, testPassword, pwLength * sizeof(char));
        }
        free(testPassword);
    }

}

char** parseFile(char* filePath, int* num, int pwLength)
{
    char* line = (char*)malloc(pwLength * sizeof(char));
    FILE* fp = fopen(filePath, "r");
    int nItems;

    fscanf(fp, "%d", &nItems);
    
    char** items = (char**)malloc(nItems * sizeof(char*));
    int i = 0;

    while (fscanf(fp, "%s", line) != EOF)
    {
        items[i] = (char*)malloc(pwLength * sizeof(char));
        strcpy(items[i], line);
        i++;
    }

    fclose(fp);
    *num = nItems;
    return items;
}

int main(int argc, char *argv[])
{
    // Arguments
    int pwLength = atoi(argv[1]);
    int mode = atoi(argv[2]);
    int threadNum = atoi(argv[3]);
    int blockNum = atoi(argv[4]);

    char *chars;
    int charsLength;
    char passwordsFilePath[100];
    char temp[2];
    strcpy(passwordsFilePath, "./passwords/");
    switch (mode)
    {
    case 0:
        chars = CHARS_MODE_0;
        charsLength = LENGTH_MODE_0;
        temp[0] = pwLength + '0';
        temp[1] = '\0';
        strcat(passwordsFilePath, "digits_passwords_");
        strcat(passwordsFilePath, temp);
        strcat(passwordsFilePath, ".txt");
        break;
    case 1:
        chars = CHARS_MODE_1;
        charsLength = LENGTH_MODE_1;
        temp[0] = pwLength + '0';
        temp[1] = '\0';
        strcat(passwordsFilePath, "lc_alphanum_passwords_");
        strcat(passwordsFilePath, temp);
        strcat(passwordsFilePath, ".txt");
        break;
    case 2:
        chars = CHARS_MODE_2;
        charsLength = LENGTH_MODE_2;
        temp[0] = pwLength + '0';
        temp[1] = '\0';
        strcat(passwordsFilePath, "alphanum_passwords_");
        strcat(passwordsFilePath, temp);
        strcat(passwordsFilePath, ".txt");
        break;
    case 3:
        chars = CHARS_MODE_3;
        charsLength = LENGTH_MODE_3;
        temp[0] = pwLength + '0';
        temp[1] = '\0';
        strcat(passwordsFilePath, "all_passwords_");
        strcat(passwordsFilePath, temp);
        strcat(passwordsFilePath, ".txt");
        break;
    }

    // Set up the test password list
    int nPasswords;
    char **passwords = parseFile(passwordsFilePath, &nPasswords, pwLength);
    char* cudaTruePassword, * cudaChars;
    int *found = (int*)malloc(sizeof(int));
    int *cudaFound;

    hipMalloc((void**)&cudaChars, charsLength * sizeof(char));
    hipMemcpy(cudaChars, chars, charsLength * sizeof(char), hipMemcpyHostToDevice);
    hipMalloc((void**)&cudaTruePassword, pwLength * sizeof(char));
    hipMalloc((void**)&cudaFound, sizeof(int));  

    // TO REMOVE
    nPasswords = 1000;
    char* foundPw, *cudaFoundPw;
    foundPw = (char*)malloc(pwLength * sizeof(char));
    hipMalloc((void**)&cudaFoundPw, pwLength * sizeof(char));
    // TO REMOVE
    double timeSum = 0.0;
    for (int i = 0; i < nPasswords; i++)
    {
        *found = 0;
        hipMemcpy(cudaFound, found, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(cudaTruePassword, passwords[i], pwLength * sizeof(char), hipMemcpyHostToDevice);

        GpuTimer timer = GpuTimer();
        timer.Start();
        bruteForce <<<blockNum, threadNum >>> (cudaTruePassword, pwLength, cudaChars, charsLength, cudaFound, cudaFoundPw, (long)pow(charsLength, pwLength));
        hipDeviceSynchronize();
        timer.Stop();
        timeSum += timer.Elapsed() / 1000.0;
        double currAvg = timeSum / (i + 1);
        hipMemcpy(found, cudaFound, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(foundPw, cudaFoundPw, pwLength * sizeof(char), hipMemcpyDeviceToHost);
        printf("[%d/%d][avg: %f]: It took %fs to find %s\n", (i + 1), nPasswords, currAvg, timer.Elapsed() / 1000.0, passwords[i]);
        printf("Actually found %s\n", foundPw);
        if (*found == 0)
        {
            printf("Failure\n");
        }
    }
    printf("Average time of %f\n", timeSum / nPasswords);
}
